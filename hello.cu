
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void Hello(){
    printf("Hello GPU from the threadId: %d\n",threadIdx.x);
}
int main(){
    Hello<<<1, 5>>>();
    hipDeviceSynchronize();
    return 0;
}
